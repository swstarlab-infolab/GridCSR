#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <array>

#include "common.h"


#ifndef CUDA_BLOCKS
#define CUDA_BLOCKS 160
#endif

#ifndef CUDA_THREADS 
#define CUDA_THREADS 1024
#endif

#ifndef CUDA_STREAMS 
#define CUDA_STREAMS 4
#endif

#define CUDA_CHECK \
        do { } while (false)
        //do { std::cout << "line: " << __LINE__ << " " << hipGetLastError() << std::endl; } while (false)

using count_t = unsigned long long;

__device__ static void bitmap_set(uint32_t* bm, const vertex_t vid) {
	atomicOr(&bm[vid >> 5], 1 << (vid & 31));
}

__device__ static bool bitmap_check(uint32_t* bm, const vertex_t vid) {
	return bm[vid >> 5] & (1 << (vid & 31));
}

__global__ static void kernel(
    vertex_t const * G0Row, vertex_t const * G0Ptr, vertex_t const * G0Col,
    vertex_t const * G1Row, vertex_t const * G1Ptr, vertex_t const * G1Col,
    vertex_t const * G2Row, vertex_t const * G2Ptr, vertex_t const * G2Col,
    count_t const G0RowSz,
    count_t const G1RowSz,
    count_t const G2RowSz,
    uint32_t * bitmap,
    count_t * count
) {
    uint32_t * mybm = &bitmap[(FORMAT_GRID_WIDTH >> 5) * blockIdx.x];
    count_t mycount = 0;

    __shared__ int SHARED[CUDA_THREADS];

    for (uint32_t g1row = blockIdx.x; g1row < FORMAT_GRID_WIDTH; g1row += gridDim.x) {
        auto const g1col_idx_s = G1Ptr[g1row];
        auto const g1col_idx_e = G1Ptr[g1row+1];

        if (g1col_idx_s == g1col_idx_e) { continue; }

        // generate bitmap
        for (uint32_t g1col_idx = g1col_idx_s + threadIdx.x; g1col_idx < g1col_idx_e; g1col_idx += blockDim.x) {
            bitmap_set(mybm, G1Col[g1col_idx]);
        }

        __syncthreads();

        auto g2col_s = G2Ptr[g1row];
        auto g2col_e = G2Ptr[g1row+1];

        for (uint32_t g2col_idx = g2col_s; g2col_idx < g2col_e; g2col_idx += blockDim.x) {
            SHARED[threadIdx.x] = (g2col_idx + threadIdx.x < g2col_e) ? (int)G2Col[g2col_idx + threadIdx.x] : -1;

            __syncthreads();

            for (uint32_t s = 0; s < blockDim.x; s++) {
                int const g2col = SHARED[s];
                if (g2col == -1) { break; }

                auto const g0col_idx_s = G0Ptr[g2col];
                auto const g0col_idx_e = G0Ptr[g2col+1];

                for (uint32_t g0col_idx = g0col_idx_s + threadIdx.x; g0col_idx < g0col_idx_e; g0col_idx += blockDim.x) {
                    if (bitmap_check(mybm, G0Col[g0col_idx])) {
                        mycount++;
                    }
                }
            }
            __syncthreads();
        }

        // reset bitmap
        for (uint32_t g1col_idx = g1col_idx_s + threadIdx.x; g1col_idx < g1col_idx_e; g1col_idx += blockDim.x) {
            mybm[G1Col[g1col_idx] >> 5] = 0;
        }

        __syncthreads();
    }

    //atomicAdd(count, mycount);

    for (uint8_t offset = 16; offset > 0; offset >>= 1) {
		mycount += __shfl_down_sync(0xFFFFFFFF, mycount, offset);
	}

	if ((threadIdx.x & 31) == 0) { atomicAdd(count, mycount); }
}

void launch(std::vector<gridInfo_t> const & info, std::vector<gridData_t> const & data) {
    //std::cout << ">>> Launch GPU" << std::endl;
    std::cout << "STREAMS: " << CUDA_STREAMS << ", BLOCKS: " << CUDA_BLOCKS << ", THREADS: " << CUDA_THREADS << std::endl;

    auto rows = info.back().pos.row + 1;
    auto cols = info.back().pos.col + 1;
    auto rc2i = [&cols](vertex_t const row, vertex_t const col) ->vertex_t{ return row * cols + col; };

    std::vector<std::vector<vertex_t*>> dRow, dPtr, dCol;

    dRow.resize(rows);
    for (auto & e : dRow) { e.resize(cols); }

    dPtr.resize(rows);
    for (auto & e : dPtr) { e.resize(cols); }

    dCol.resize(rows);
    for (auto & e : dCol) { e.resize(cols); }

    hipDeviceReset(); CUDA_CHECK;
    hipSetDevice(0); CUDA_CHECK;

    std::array<count_t *, CUDA_STREAMS> dcount;
    for (uint32_t i = 0; i < CUDA_STREAMS; i++) {
        hipMalloc(&dcount[i], sizeof(count_t)); CUDA_CHECK;
        hipMemset(dcount[i], 0, sizeof(count_t)); CUDA_CHECK;
    }

    std::array<uint32_t *, CUDA_STREAMS> bitmap;
    for (uint32_t i = 0; i < CUDA_STREAMS; i++) {
        hipMalloc(&bitmap[i], sizeof(uint32_t) * (FORMAT_GRID_WIDTH / 32) * CUDA_BLOCKS); CUDA_CHECK;
        hipMemset(bitmap[i], 0, sizeof(uint32_t) * (FORMAT_GRID_WIDTH / 32) * CUDA_BLOCKS); CUDA_CHECK;
    }

    //std::cout << "complete: GPU bitmap malloc & memset" << std::endl;

    for (uint32_t row = 0; row < rows; row++) {
        for (uint32_t col = 0; col <= row; col++) {
            auto idx = rc2i(row, col);
            //std::cout << "   Malloc Grid: " << row << ", " << col << std::endl;
            hipMalloc(&dRow[row][col], data[idx].row.size() * sizeof(vertex_t)); CUDA_CHECK;
            hipMalloc(&dPtr[row][col], data[idx].ptr.size() * sizeof(vertex_t)); CUDA_CHECK;
            hipMalloc(&dCol[row][col], data[idx].col.size() * sizeof(vertex_t)); CUDA_CHECK;
        }
    }

    //std::cout << "complete: GPU graph data malloc" << std::endl;

    for (uint32_t row = 0; row < rows; row++) {
        for (uint32_t col = 0; col <= row; col++) {
            auto idx = rc2i(row, col);
            //std::cout << "   Copy Grid: " << row << ", " << col << std::endl;
            hipMemcpy(dRow[row][col], data[idx].row.data(), data[idx].row.size() * sizeof(vertex_t), hipMemcpyHostToDevice); CUDA_CHECK;
            hipMemcpy(dPtr[row][col], data[idx].ptr.data(), data[idx].ptr.size() * sizeof(vertex_t), hipMemcpyHostToDevice); CUDA_CHECK;
            hipMemcpy(dCol[row][col], data[idx].col.data(), data[idx].col.size() * sizeof(vertex_t), hipMemcpyHostToDevice); CUDA_CHECK;
        }
    }

    //std::cout << "complete: GPU graph data memcpy" << std::endl;

    std::array<hipStream_t, CUDA_STREAMS> stream;
    for (uint32_t i = 0; i < CUDA_STREAMS; i++) {
        hipStreamCreate(&stream[i]); CUDA_CHECK;
    }

    //std::cout << "complete: GPU stream create" << std::endl;

    std::array<count_t, CUDA_STREAMS> count = {0, };

    auto start = std::chrono::system_clock::now();


    uint32_t stream_number = 0;
    for (uint32_t row = 0; row < rows; row++) {
        for (uint32_t col = 0; col <= row; col++) {
            for (uint32_t i = col; i <= row; i++) {
                kernel <<<CUDA_BLOCKS, CUDA_THREADS, 0, stream[stream_number]>>> (
                    dRow[i][col], dPtr[i][col],   dCol[i][col],
                    dRow[row][col], dPtr[row][col], dCol[row][col],
                    dRow[row][i], dPtr[row][i],   dCol[row][i],
                    data[rc2i(i, col)].row.size(),
                    data[rc2i(row, col)].row.size(),
                    data[rc2i(row, i)].row.size(),
                    bitmap[stream_number],
                    dcount[stream_number]
                );

                stream_number++;
                if (stream_number / CUDA_STREAMS != 0) {
                    stream_number = 0;
                }
            }
        }
    }

    for (uint32_t i = 0; i < CUDA_STREAMS; i++) {
        hipMemcpyAsync(&count[i], dcount[i], sizeof(count_t), hipMemcpyDeviceToHost, stream[i]); CUDA_CHECK;
        hipStreamSynchronize(stream[i]); CUDA_CHECK;
    }

    for (uint32_t i = 1; i < CUDA_STREAMS; i++) {
        count[0] += count[i];
    }

    std::chrono::duration<double> elapsed = std::chrono::system_clock::now() - start;
    std::cout << count[0] << "," << elapsed.count() << std::endl;

    for (uint32_t i = 0; i < CUDA_STREAMS; i++) {
        hipStreamDestroy(stream[i]); CUDA_CHECK;
    }

    hipDeviceReset(); CUDA_CHECK;
}