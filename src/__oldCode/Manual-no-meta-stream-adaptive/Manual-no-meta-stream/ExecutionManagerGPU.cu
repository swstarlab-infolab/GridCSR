#include "hip/hip_runtime.h"
#include "ExecutionManager.cuh"
#include "type.cuh"

#include <GridCSR/CUDA/Kernel.cuh>
#include <cub/device/device_scan.cuh>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

#define CUDACHECK()                        \
	do {                                   \
		auto e = hipGetLastError();       \
		if (e) {                           \
			printf("%s:%d, %s(%d), %s\n",  \
				   __FILE__,               \
				   __LINE__,               \
				   hipGetErrorName(e),    \
				   e,                      \
				   hipGetErrorString(e)); \
			hipDeviceReset();             \
			exit(EXIT_FAILURE);            \
		}                                  \
	} while (false)

static __global__ void genLookupTemp(Grid const g, Lookup * luTemp)
{
	for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < g[0].count();
		 i += gridDim.x * blockDim.x) {
		luTemp[g[0][i]] = g[1][i + 1] - g[1][i];
	}
}

static __global__ void resetLookupTemp(Grid const g, Lookup * luTemp)
{
	for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < g[0].count();
		 i += gridDim.x * blockDim.x) {
		luTemp[g[0][i]] = 0;
	}
}

static __global__ void
kernel(Grids const g, Lookup const * lookup0, Lookup const * lookup2, Count * count)
{
	Count mycount = 0;

	__shared__ int SHARED[1024];

	for (uint32_t g1row_iter = blockIdx.x; g1row_iter < g[1][0].count(); g1row_iter += gridDim.x) {

		// This makes huge difference!!!
		// Without "Existing Row" information: loop all 2^24 and check it all
		// With "Existing Row" information: extremely faster than without-version
		auto const g1row = g[1][0][g1row_iter];

		if (lookup2[g1row] == lookup2[g1row + 1]) {
			continue;
		}

		auto const g1col_idx_s = g[1][1][g1row_iter];
		auto const g1col_idx_e = g[1][1][g1row_iter + 1];

		// variable for binary tree intersection
		auto const g1col_length = g1col_idx_e - g1col_idx_s;

		auto const g2col_s = lookup2[g1row], g2col_e = lookup2[g1row + 1];

		for (uint32_t g2col_idx = g2col_s; g2col_idx < g2col_e; g2col_idx += blockDim.x) {
			SHARED[threadIdx.x] =
				(g2col_idx + threadIdx.x < g2col_e) ? (int)g[2][2][g2col_idx + threadIdx.x] : -1;

			__syncthreads();

			for (uint32_t s = 0; s < blockDim.x; s++) {
				int const g2col = SHARED[s];
				if (g2col == -1) {
					break;
				}
				if (lookup0[g2col] == lookup0[g2col + 1]) {
					continue;
				}

				auto const g0col_idx_s = lookup0[g2col], g0col_idx_e = lookup0[g2col + 1];

				// variable for binary tree intersection
				auto const g0col_length = g0col_idx_e - g0col_idx_s;

				if (g1col_length >= g0col_length) {
					for (uint32_t g0col_idx = g0col_idx_s + threadIdx.x; g0col_idx < g0col_idx_e;
						 g0col_idx += blockDim.x) {
						GridCSR::CUDA::BinarySearchIntersection(
							&g[1][2][g1col_idx_s], g1col_length, g[0][2][g0col_idx], &mycount);
					}
				} else {
					for (uint32_t g1col_idx = g1col_idx_s + threadIdx.x; g1col_idx < g1col_idx_e;
						 g1col_idx += blockDim.x) {
						GridCSR::CUDA::BinarySearchIntersection(
							&g[0][2][g0col_idx_s], g0col_length, g[1][2][g1col_idx], &mycount);
					}
				}
			}
			__syncthreads();
		}
	}

	for (uint8_t offset = 16; offset > 0; offset >>= 1) {
		mycount += __shfl_down_sync(0xFFFFFFFF, mycount, offset);
	}

	if ((threadIdx.x & 31) == 0) {
		atomicAdd(count, mycount);
	}
}

Count launchKernelGPU(Context & ctx, DeviceID myID, size_t myStreamID, Grids & G)
{

	auto & myCtx   = ctx.executionManagerCtx[myID].my[myStreamID];
	auto & blocks  = ctx.setting[1];
	auto & threads = ctx.setting[2];

	auto stream = myCtx.stream;

	if (!(G[0][0].byte && G[1][0].byte && G[2][0].byte)) {
		return 0;
	}

	hipSetDevice(myID);
	hipMemsetAsync(myCtx.count.ptr, 0x00, myCtx.count.byte, stream);
	CUDACHECK();

	hipSetDevice(myID);
	genLookupTemp<<<blocks, threads, 0, stream>>>(G[0], myCtx.lookup.temp.ptr);
	CUDACHECK();

	hipSetDevice(myID);
	hipcub::DeviceScan::ExclusiveSum(myCtx.cub.ptr,
								  myCtx.cub.byte,
								  myCtx.lookup.temp.ptr,
								  myCtx.lookup.G0.ptr,
								  myCtx.lookup.G0.count(),
								  stream);
	CUDACHECK();

	hipSetDevice(myID);
	resetLookupTemp<<<blocks, threads, 0, stream>>>(G[0], myCtx.lookup.temp.ptr);
	CUDACHECK();

	hipSetDevice(myID);
	genLookupTemp<<<blocks, threads, 0, stream>>>(G[2], myCtx.lookup.temp.ptr);
	CUDACHECK();

	hipSetDevice(myID);
	hipcub::DeviceScan::ExclusiveSum(myCtx.cub.ptr,
								  myCtx.cub.byte,
								  myCtx.lookup.temp.ptr,
								  myCtx.lookup.G2.ptr,
								  myCtx.lookup.G2.count(),
								  stream);
	CUDACHECK();

	hipSetDevice(myID);
	resetLookupTemp<<<blocks, threads, 0, stream>>>(G[2], myCtx.lookup.temp.ptr);
	CUDACHECK();

	hipSetDevice(myID);
	kernel<<<blocks, threads, 0, stream>>>(
		G, myCtx.lookup.G0.ptr, myCtx.lookup.G2.ptr, myCtx.count.ptr);
	CUDACHECK();

	Count cnt = 0;
	hipSetDevice(myID);
	hipMemcpyAsync(&cnt, myCtx.count.ptr, sizeof(Count), hipMemcpyDeviceToHost, stream);
	CUDACHECK();

	hipStreamSynchronize(stream);
	CUDACHECK();

	return cnt;
}
// Under construction...
