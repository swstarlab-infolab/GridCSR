#include "main.cuh"

#include <fstream>
#include <string>
#include <sstream>
#include "../meta.h"

static size_t filesize(fs::path const & path) {
    std::ifstream f;
    f.open(path);
    f.seekg(0, std::ios::end);
    auto const fileSize = f.tellg();
    f.seekg(0, std::ios::beg);
    f.close();
    return fileSize;
}

static std::string SI(size_t byte) {
    size_t constexpr KiB = size_t(1) << 10;
    size_t constexpr MiB = size_t(1) << 20;
    size_t constexpr GiB = size_t(1) << 30;
    size_t constexpr TiB = size_t(1) << 40;
    size_t constexpr PiB = size_t(1) << 50;
    size_t constexpr EiB = size_t(1) << 60;
    char buf[11];
    if (byte < KiB) {
        snprintf(buf, sizeof(buf), "%ldB", byte);
    } else if (KiB <= byte && byte < MiB) {
        snprintf(buf, sizeof(buf), "%.3lfKiB", double(byte) / double(KiB));
    } else if (MiB <= byte && byte < GiB) {
        snprintf(buf, sizeof(buf), "%.3lfMiB", double(byte) / double(MiB));
    } else if (GiB <= byte && byte < TiB) {
        snprintf(buf, sizeof(buf), "%.3lfGiB", double(byte) / double(GiB));
    } else if (TiB <= byte && byte < PiB) {
        snprintf(buf, sizeof(buf), "%.3lfTiB", double(byte) / double(TiB));
    } else if (PiB <= byte && byte < EiB) {
        snprintf(buf, sizeof(buf), "%.3lfPiB", double(byte) / double(PiB));
    } else {
        snprintf(buf, sizeof(buf), "%.3lfEiB", double(byte) / double(EiB));
    }

    std::ostringstream oss;
    oss << buf;
    std::string result = oss.str();
    return result;
}

static hipError_t loadFileToGPU(
    fs::path const & path,
    CudaMemory<vertex_t> & mem)
{
    auto fileSize = filesize(path);

    std::vector<vertex_t> temp(fileSize / sizeof(vertex_t));

    std::ifstream f;
    f.open(path, std::ios::binary);
    f.read((char*)temp.data(), fileSize);
    printf("%s, %ld\n", path.c_str(), fileSize / sizeof(vertex_t));
    f.close();

    mem.mallocByte(fileSize);
    auto e = hipGetLastError(); if (e) { return e; }
    mem.copyH2D(temp.data()); CUDACHECK();
    temp.clear();

    return hipError_t::hipSuccess;
}

int selectGPU(int const devices) {
    size_t maxfree = 0;
    size_t maxgpu = 0;
    for (int i = 0; i < devices; i++) {
        size_t free, total;
        hipSetDevice(i);
        hipDeviceSynchronize();
        hipMemGetInfo(&free, &total);
        if (maxfree < free) {
            maxfree = free;
            maxgpu = i;
        }
    }
    return maxgpu;
}

void managed_t::init(fs::path const & folderPath) {
    meta_t meta;
    meta.unmarshal_from_file(fs::path(folderPath.string() + "meta.json"));

    std::ifstream f;
    this->graph.resize(meta.info.count.row);
    for (auto & g : this->graph) {
        g.resize(meta.info.count.col);
    }

    int devices = -1;
    hipGetDeviceCount(&devices);

    for (auto i = 0; i < meta.grid.each.size(); i++) {
        auto const baseString = folderPath.string() + std::string(meta.grid.each[i].name) + ".";

        auto const pathRow = fs::path(baseString + meta.extension.row);
        auto const pathPtr = fs::path(baseString + meta.extension.ptr);
        auto const pathCol = fs::path(baseString + meta.extension.col);

        if (!(fs::exists(pathRow) && fs::exists(pathPtr) && fs::exists(pathCol))) {
            printf("Not exists: %s\n", meta.grid.each[i].name.c_str());
            exit(EXIT_FAILURE);
        }

        size_t const rowIndex = meta.grid.each[i].index.row;
        size_t const colIndex = meta.grid.each[i].index.col;

        int gpuID = selectGPU(devices);
        printf("GPU: %d\n", gpuID);

        hipSetDevice(gpuID);

        while (true) {
            auto e = loadFileToGPU(pathRow, this->graph[rowIndex][colIndex].row);
            if (e == hipErrorOutOfMemory) {
                exit(-1);
            } else if (e == hipSuccess) {
                break;
            } else {
                fprintf(stderr, "error, %s\n", hipGetErrorString(e)); exit(-1);
            }
        }

        while (true) {
            auto e = loadFileToGPU(pathPtr, this->graph[rowIndex][colIndex].ptr);
            if (e == hipErrorOutOfMemory) {
                exit(-1);
            } else if (e == hipSuccess) {
                break;
            } else {
                fprintf(stderr, "error, %s\n", hipGetErrorString(e)); exit(-1);
            }
        }

        while (true) {
            auto e = loadFileToGPU(pathCol, this->graph[rowIndex][colIndex].col);
            if (e == hipErrorOutOfMemory) {
                exit(-1);
            } else if (e == hipSuccess) {
                break;
            } else {
                fprintf(stderr, "error, %s\n", hipGetErrorString(e)); exit(-1);
            }
        } 
    }

    for (int i = 0; i < devices; i++) {
        size_t free, total;
        hipSetDevice(i);
        hipMemGetInfo(&free, &total);
        fprintf(stdout, "GPU %d : %s / %s\n", i, SI(free).c_str(), SI(total).c_str());
    }

    hipDeviceSynchronize();
}