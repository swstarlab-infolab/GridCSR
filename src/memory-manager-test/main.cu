#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <algorithm>
#include <cstdint>

#include <cstdio>
#define CUDACHECK() \
        do { auto e = hipGetLastError(); if (e) { printf("%s:%d, %s(%d), %s\n", __FILE__, __LINE__, hipGetErrorName(e), e , hipGetErrorString(e)); hipDeviceReset(); exit(EXIT_FAILURE); } } while (false)


using count_t = unsigned long long;

__global__
void kernel(int * pointer, size_t size, count_t * out) {
    count_t mycount = 0;
    int TID = blockDim.x * blockIdx.x + threadIdx.x;
    int TS = gridDim.x * blockDim.x;

    for (int i = TID; i < size; i+=TS) {
        mycount++;
    }

    for (uint8_t offset = 16; offset > 0; offset >>= 1) {
		mycount += __shfl_down_sync(0xFFFFFFFF, mycount, offset);
	}

	if ((threadIdx.x & 31) == 0) { atomicAdd(out, mycount); }
}

int main() {
    int * a = nullptr;
    count_t * b = nullptr;
    size_t tile_size = (128 * 1024 * 1024) / sizeof(*a);
    size_t num_tiles = 1000;

    hipMallocManaged(&a, tile_size * num_tiles * sizeof(*a)); CUDACHECK();

    hipSetDevice(0);

    hipMalloc(&b, sizeof(*b));
    hipMemset(b, 0, sizeof(*b));

    hipStream_t s[2];
    hipStreamCreate(&s[0]); CUDACHECK();
    hipStreamCreate(&s[1]); CUDACHECK();

    hipEvent_t e[2];
    hipEventCreate(&e[0]); CUDACHECK();
    hipEventCreate(&e[1]); CUDACHECK();

    hipMemPrefetchAsync(&a[0], tile_size * sizeof(*a), 0, s[1]); CUDACHECK();
    hipEventRecord(e[0], s[1]);  CUDACHECK();

    for (int i = 0; i < num_tiles; i++) { 
        hipEventSynchronize(e[0]); CUDACHECK();
        hipEventSynchronize(e[1]); CUDACHECK();

        kernel<<<160, 1024, 0, s[0]>>>(&a[tile_size * i], tile_size, b); 
        hipEventRecord(e[0], s[0]); CUDACHECK(); 

        if (i < num_tiles-1) {
            hipMemPrefetchAsync(&a[tile_size * (i+1)], tile_size * sizeof(*a), 0, s[1]); CUDACHECK();
            hipEventRecord(e[1], s[1]); CUDACHECK();
        } 

        std::swap(s[0], s[1]);
        std::swap(e[0], e[1]);
    }

    count_t hb = 0;
    hipMemcpy(&hb, b, sizeof(*b), hipMemcpyDeviceToHost);

    printf("hb: %lld\n", hb);

    hipDeviceReset(); CUDACHECK();

    return 0;
}