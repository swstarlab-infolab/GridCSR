#include <cstdio>
#include <vector>

#include <hip/hip_runtime.h>


#define bufSize 16

__global__ void kernel(float * src, float * dst) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int ts = gridDim.x * blockDim.x;

    for (int i = tid; i < bufSize; i += ts) {
        dst[i] = src[i] * 2.0f;
    }
}

int main() {
    int GPUs;
    hipGetDeviceCount(&GPUs);

    if (GPUs < 2) {
        return 0;
    }

    std::vector<hipDeviceProp_t> prop(GPUs);
    for (int i = 0; i < GPUs; i++) {
        hipGetDeviceProperties(&prop[i], i);
    }

    for (int i = 0; i < GPUs; i++) {
        for (int j = 0; j < GPUs; j++) {
            if (i == j) { continue; }
            int canAccess;
            hipDeviceCanAccessPeer(&canAccess, i, j);
            if (canAccess) {
                hipSetDevice(i);
                hipDeviceEnablePeerAccess(j, 0);
            }
        }
    }

    std::vector<float*> g(GPUs);
    std::vector<float*> h(1);

    for (int i = 0; i < g.size(); i++) {
        hipSetDevice(i);
        hipMalloc(&g[i], sizeof(float) * bufSize);
    }

    for (int i = 0; i < h.size(); i++) {
        hipSetDevice(i);
        hipHostMalloc(&h[i], sizeof(float) * bufSize, hipHostMallocDefault);
    }

    for (int i = 0; i < bufSize; i++) {
        h[0][i] = (float)i;
    }

    for (int i = 0; i < bufSize; i++) {
        printf("%f ", h[0][i]);
    }
    printf("\n");

    hipMemcpy(g[0], h[0], sizeof(float) * bufSize, hipMemcpyHostToDevice);

    for (int i = 0; i < GPUs; i++) {
        if (i != GPUs - 1) {
            kernel<<<1, 32>>>(g[i], g[i+1]);
        } else {
            kernel<<<1, 32>>>(g[i], g[0]);
        }
    }

    hipMemcpy(h[0], g[0], sizeof(float) * bufSize, hipMemcpyDeviceToHost);

    for (int i = 0; i < bufSize; i++) {
        printf("%f ", h[0][i]);
    }
    printf("\n");

    hipDeviceReset();
    return 0; 
 }