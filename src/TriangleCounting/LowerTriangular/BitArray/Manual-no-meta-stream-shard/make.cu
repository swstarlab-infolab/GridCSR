#include "make.cuh"

template <>
std::shared_ptr<void> allocCUDA<void>(size_t const byte)
{
	return std::shared_ptr<void>(
		[&] {
			void * p;
			hipMalloc((void **)&p, byte);
			return p;
		}(),
		[](void * p) {
			if (p != nullptr) {
				hipFree(p);
			}
		});
}

template <>
std::shared_ptr<void> allocHost<void>(size_t const byte)
{
	return std::shared_ptr<void>(
		[&] {
			void * p;
			hipHostAlloc((void **)&p, byte, hipHostMallocPortable);
			return p;
		}(),
		[](void * p) {
			if (p != nullptr) {
				hipFree(p);
			}
		});
}