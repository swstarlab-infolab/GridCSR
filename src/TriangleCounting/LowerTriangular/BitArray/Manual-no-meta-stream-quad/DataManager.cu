#include "DataManager.cuh"
#include "make.cuh"

#include <BuddySystem/BuddySystem.h>
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <fstream>
#include <limits>
#include <mutex>
#include <string>
#include <tuple>
#include <unistd.h>
#include <unordered_map>

static auto genPath(Context & ctx, Key const & k)
{
	auto baseString =
		std::string(ctx.folderPath) + std::to_string(k.idx[0]) + "-" + std::to_string(k.idx[1]);

	fs::path finalPath;
	switch (k.type) {
	case DataType::Row:
		finalPath = fs::path(baseString + ".row");
		break;
	case DataType::Ptr:
		finalPath = fs::path(baseString + ".ptr");
		break;
	case DataType::Col:
		finalPath = fs::path(baseString + ".col");
		break;
	}

	return finalPath;
}

static auto methodDone(Context & ctx, DeviceID myID)
{
	auto in = std::make_shared<bchan<Tx>>(16);
	std::thread([&, myID, in] {
		auto & myCtx = ctx.dataManagerCtx[myID];
		for (auto & tx : *in) {
			MemInfo<Vertex> myInfo = {
				0,
			};
			{
				std::lock_guard<std::mutex> lg(*myCtx.cacheMtx);
				myCtx.cache->at(tx.key).refCnt -= 1;
				// printf("[%2d] %s Done %d -> %d\n", myID, tx.key.print().c_str(),
				// myCtx.cache->at(tx.key).refCnt + 1, myCtx.cache->at(tx.key).refCnt);
			}
			myInfo.ok  = true;
			myInfo.hit = true;
			tx.cb->push(myInfo);
			tx.cb->close();
		}
	}).detach();

	return in;
}

static void tryAllocate(Context &					   ctx,
						Key &						   key,
						DeviceID					   myID,
						MemInfo<Vertex> &			   myInfo,
						std::unique_lock<std::mutex> & ul,
						bool &						   iHaveLock)
{
	auto & myCtx = ctx.dataManagerCtx[myID];

	while (true) {
		myInfo.ptr = (Vertex *)myCtx.buddy->allocate(myInfo.byte);

		if (myInfo.ptr != nullptr) {
			myCtx.cache->insert({key, {myInfo, 1}});
			// printf("[%2d] %s Allc %d -> %d\n", myID, key.print().c_str(),
			// myCtx.cache->at(key).refCnt - 1, myCtx.cache->at(key).refCnt);

			myInfo.ok  = true;
			myInfo.hit = false;

			return;
		} else {
			// allocation failure
			if (myCtx.cache->size() > 0) {
				bool evictSuccess = false;
				while (!evictSuccess) {
					if (!iHaveLock) {
						ul.lock();
						iHaveLock = true;
					}

					for (auto it = myCtx.cache->begin(); it != myCtx.cache->end();) {
						if (it->second.refCnt == 0) {
							myCtx.buddy->deallocate(it->second.info.ptr);
							it			 = myCtx.cache->erase(it);
							evictSuccess = true;
							// printf("[%2d] %s Evict %s\n", myID, key.print().c_str(),
							// it->first.print().c_str());
							break;
						} else {
							++it;
						}
					}

					if (iHaveLock) {
						ul.unlock();
						iHaveLock = false;
					}
				}
			} else {
				throw "Strange Error";
			}
		}
	}
}

static MemInfo<Vertex> requestToReady(Context & ctx, Key & key, DeviceID targetID)
{
	Tx tx;

	tx.key	  = key;
	tx.method = Method::Ready;
	tx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

	ctx.dataManagerCtx[targetID].chan->push(tx);

	MemInfo<Vertex> otherInfo;
	for (auto & info : *tx.cb) { // Code hangs on this line
		otherInfo = info;
	}

	return otherInfo;
}

static auto methodReady(Context & ctx, DeviceID myID)
{
	auto in = std::make_shared<bchan<Tx>>(16);
	std::thread([&, myID, in] {
		for (auto & tx : *in) {
			auto & myCtx = ctx.dataManagerCtx[myID];

			MemInfo<Vertex> myInfo = {
				0,
			};

			std::unique_lock<std::mutex> ul(*myCtx.cacheMtx);

			bool iHaveLock = true;

			if (myCtx.cache->find(tx.key) != myCtx.cache->end()) {
				// printf("[%2d] %s Hit!\n", myID, tx.key.print().c_str());
				myInfo = myCtx.cache->at(tx.key).info;
				myCtx.cache->at(tx.key).refCnt += 1;
				// printf("[%2d] %s Hit  %d -> %d\n", myID, tx.key.print().c_str(),
				// myCtx.cache->at(tx.key).refCnt - 1, myCtx.cache->at(tx.key).refCnt);

				if (iHaveLock) {
					ul.unlock();
					iHaveLock = false;
				}

				myInfo.hit = true;
			} else {
				// printf("[%2d] %s Miss!\n", myID, tx.key.print().c_str());

				auto exts		= std::array<std::string, 3>{".row", ".ptr", ".col"};
				auto targetPath = fs::path((ctx.folderPath / fs::path(tx.key.idx)).string() +
										   exts[(size_t)tx.key.type]);
				myInfo.byte		= fs::file_size(targetPath);

				tryAllocate(ctx, tx.key, myID, myInfo, ul, iHaveLock);

				assert(myID >= -1);
				if (myID == -1) {
					// printf("start to read!\n");

					// CPU
					// std::ifstream f(otherInfo.path, std::ios::binary);
					// printf("[%2d] %s fread       SSD[%s]->Host[%p], %ld bytes)\n", myID,
					// tx.key.print().c_str(), otherInfo.path.c_str(), myInfo.ptr,
					// otherInfo.byte);

					auto fp = open64(targetPath.c_str(), O_RDONLY);

					constexpr uint64_t cDef		 = (1L << 30); // chunk Default
					uint64_t		   chunkByte = (myInfo.byte < cDef) ? myInfo.byte : cDef;
					uint64_t		   bytePos	 = 0;
					while (bytePos < myInfo.byte) {
						chunkByte =
							(myInfo.byte - bytePos > chunkByte) ? chunkByte : myInfo.byte - bytePos;
						auto loaded = read(fp, &(((uint8_t *)myInfo.ptr)[bytePos]), chunkByte);
						bytePos += loaded;
					}

					close(fp);
				} else {
					auto otherInfo = requestToReady(ctx, tx.key, myCtx.conn->upstream);
					// GPU
					// printf("[%2d] %s hipMemcpy Host[%p]-> GPU[%p], %ld bytes)\n", myID,
					// tx.key.print().c_str(), otherInfo.ptr, myInfo.ptr, otherInfo.byte);
					hipSetDevice(myID);
					hipMemcpyAsync(myInfo.ptr,
									otherInfo.ptr,
									otherInfo.byte,
									hipMemcpyHostToDevice,
									myCtx.stream);
					hipStreamSynchronize(myCtx.stream);

					// Done
					Tx compTx;

					compTx.key	  = tx.key;
					compTx.method = Method::Done;
					compTx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

					ctx.dataManagerCtx[myCtx.conn->upstream].chan->push(compTx);

					for (auto & res : *compTx.cb) {
					}
				}

				// printf("[%2d] %s Memcpy/Read complete\n", myID, tx.key.print().c_str());

				if (iHaveLock) {
					ul.unlock();
					iHaveLock = false;
				}

				myInfo.hit = false;
				myInfo.ok  = true;
			}

			tx.cb->push(myInfo);
			tx.cb->close();
		}
	}).detach();
	return in;
}

void DataManager(Context & ctx, DeviceID myID)
{
	// Main/GPU Memory
	std::thread([&, myID] {
		auto ReadyChan = methodReady(ctx, myID);
		auto DoneChan  = methodDone(ctx, myID);

		for (auto & tx : *ctx.dataManagerCtx[myID].chan) {
			switch (tx.method) {
			case Method::Ready:
				ReadyChan->push(tx);
				break;
			case Method::Done:
				DoneChan->push(tx);
				break;
			}
		}

		ReadyChan->close();
		DoneChan->close();
	}).detach();
}