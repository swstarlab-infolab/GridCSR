#include "hip/hip_runtime.h"
#include "ExecutionManager.cuh"

#include <tbb/blocked_range.h>
#include <tbb/parallel_scan.h>
#include <thread>

static void genLookupTemp(Grid const g, Lookup * luTemp)
{
	for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < g[0].count();
		 i += gridDim.x * blockDim.x) {
		luTemp[g[0][i]] = g[1][i + 1] - g[1][i];
	}
}

static void resetLookupTemp(Grid const g, Lookup * luTemp)
{
	for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < g[0].count();
		 i += gridDim.x * blockDim.x) {
		luTemp[g[0][i]] = 0;
	}
}

static void kernel(Grids const g, Lookup const * lookup0, Lookup const * lookup2, Count * count)
{
	Count mycount = 0;

	int SHARED[1024];

	for (uint32_t g1row_iter = blockIdx.x; g1row_iter < g[1][0].count(); g1row_iter += gridDim.x) {

		// This makes huge difference!!!
		// Without "Existing Row" information: loop all 2^24 and check it all
		// With "Existing Row" information: extremely faster than without-version
		auto const g1row = g[1][0][g1row_iter];

		if (lookup2[g1row] == lookup2[g1row + 1]) {
			continue;
		}

		auto const g1col_idx_s = g[1][1][g1row_iter];
		auto const g1col_idx_e = g[1][1][g1row_iter + 1];

		// variable for binary tree intersection
		auto const g1col_length = g1col_idx_e - g1col_idx_s;

		auto const g2col_s = lookup2[g1row], g2col_e = lookup2[g1row + 1];

		for (uint32_t g2col_idx = g2col_s; g2col_idx < g2col_e; g2col_idx += blockDim.x) {
			SHARED[threadIdx.x] =
				(g2col_idx + threadIdx.x < g2col_e) ? (int)g[2][2][g2col_idx + threadIdx.x] : -1;

			__syncthreads();

			for (uint32_t s = 0; s < blockDim.x; s++) {
				int const g2col = SHARED[s];
				if (g2col == -1) {
					break;
				}
				if (lookup0[g2col] == lookup0[g2col + 1]) {
					continue;
				}

				auto const g0col_idx_s = lookup0[g2col], g0col_idx_e = lookup0[g2col + 1];

				// variable for binary tree intersection
				auto const g0col_length = g0col_idx_e - g0col_idx_s;

				if (g1col_length >= g0col_length) {
					for (uint32_t g0col_idx = g0col_idx_s + threadIdx.x; g0col_idx < g0col_idx_e;
						 g0col_idx += blockDim.x) {
						GridCSR::CUDA::BinarySearchIntersection(
							&g[1][2][g1col_idx_s], g1col_length, g[0][2][g0col_idx], &mycount);
					}
				} else {
					for (uint32_t g1col_idx = g1col_idx_s + threadIdx.x; g1col_idx < g1col_idx_e;
						 g1col_idx += blockDim.x) {
						GridCSR::CUDA::BinarySearchIntersection(
							&g[0][2][g0col_idx_s], g0col_length, g[1][2][g1col_idx], &mycount);
					}
				}
			}
			__syncthreads();
		}
	}

	for (uint8_t offset = 16; offset > 0; offset >>= 1) {
		mycount += __shfl_down_sync(0xFFFFFFFF, mycount, offset);
	}

	if ((threadIdx.x & 31) == 0) {
		atomicAdd(count, mycount);
	}
}

Count launchKernelCPU(Context & ctx, DeviceID myID, Grids & G)
{
	auto & myCtx   = ctx.executionManagerCtx[myID];
	auto & blocks  = ctx.setting[1];
	auto & threads = ctx.setting[2];

	if (!(G[0][0].byte && G[1][0].byte && G[2][0].byte)) {
		return 0;
	}

	Count cnt = 0;

	genLookupTemp(G[0], myCtx.lookup.temp.ptr);

	// exclusive sum
	tbb::parallel_scan(
		tbb::blocked_range<>(0, myCtx.lookup.temp.ptr),
		0,
		[&](tbb::blocked_range<size_t> const & r, uint64_t sum, bool isFinalScan) {
			auto temp = sum;
			for (size_t grain = r.begin(); grain != r.end(); grain += r.grainsize()) {
				for (size_t offset = 0; offset < r.grainsize(); offset++) {
					auto i = grain + offset;
					temp += (getBit(i) ? 1 : 0);
					if (isFinalScan) {
						myCtx.lookup.G0.ptr[i + 1] = temp;
					}
				}
			});
			return temp;
		},
		[&](size_t const & l, size_t const & r) { return l + r; },
		tbb::auto_partitioner());

	// reset lookup temp
	tbb::parallel_for(
		tbb::blocked_range<size_t>(0, myCtx.lookup.temp.size()),
		[&](tbb::blocked_range<size_t> const & r) {
			for (size_t grain = r.begin(); grain != r.end(); grain += r.grainsize()) {
				for (size_t offset = 0; offset < r.grainsize(); offset++) {
					auto i = grain + offset;

					myCtx.lookup.temp.ptr[i] = 0;
				}
			}
		},
		tbb::auto_partitioner());

	genLookupTemp(G[2], myCtx.lookup.temp.ptr);

	// exclusive sum
	tbb::parallel_scan(
		tbb::blocked_range<>(0, myCtx.lookup.temp.ptr),
		0,
		[&](tbb::blocked_range<size_t> const & r, uint64_t sum, bool isFinalScan) {
			auto temp = sum;
			for (size_t grain = r.begin(); grain != r.end(); grain += r.grainsize()) {
				for (size_t offset = 0; offset < r.grainsize(); offset++) {
					auto i = grain + offset;
					temp += (getBit(i) ? 1 : 0);
					if (isFinalScan) {
						myCtx.lookup.G2.ptr[i + 1] = temp;
					}
				}
			});
			return temp;
		},
		[&](size_t const & l, size_t const & r) { return l + r; },
		tbb::auto_partitioner());

	// reset lookup temp
	tbb::parallel_for(
		tbb::blocked_range<size_t>(0, myCtx.lookup.temp.size()),
		[&](tbb::blocked_range<size_t> const & r) {
			for (size_t grain = r.begin(); grain != r.end(); grain += r.grainsize()) {
				for (size_t offset = 0; offset < r.grainsize(); offset++) {
					auto i = grain + offset;

					myCtx.lookup.temp.ptr[i] = 0;
				}
			}
		},
		tbb::auto_partitioner());

	kernel(G, myCtx.lookup.G0.ptr, myCtx.lookup.G2.ptr, myCtx.count.ptr);

	return cnt;
}