#include "ExecutionManager.h"
#include "context.h"
#include "type.h"

#include <cub/device/device_scan.cuh>
#include <hip/hip_runtime.h>
#include <exception>
#include <thread>

void ExecutionManager::init(int const id, sp<DataManager> dm)
{
	this->ID = id;
	this->DM = dm;

	if (this->ID > -1) {
		this->initGPU();
	} else if (this->ID == -1) {
		this->initCPU();
	} else {
		throw std::runtime_error("Wrong Index");
	}
}

void ExecutionManager::initCPU()
{
	for (auto & lu : this->mem.lookup) {
		lu.byte = sizeof(Vertex32) * ctx.grid.width;
		lu.ptr	= (Vertex32 *)this->DM->manualAlloc(lu.byte);
		memset(lu.ptr, 0x00, lu.byte);
	}

	this->mem.count.byte = sizeof(Count);
	this->mem.count.ptr	 = (Count *)this->DM->manualAlloc(this->mem.count.byte);
	memset(this->mem.count.ptr, 0x00, this->mem.count.byte);
}

void ExecutionManager::initGPU()
{
	for (auto & lu : this->mem.lookup) {
		hipSetDevice(this->ID);
		lu.byte = sizeof(Vertex32) * ctx.grid.width;
		lu.ptr	= (Vertex32 *)this->DM->manualAlloc(lu.byte);
		hipMemset(lu.ptr, 0x00, lu.byte);
	}

	hipSetDevice(this->ID);
	this->mem.count.byte = sizeof(Count);
	this->mem.count.ptr	 = (Count *)this->DM->manualAlloc(this->mem.count.byte);
	hipMemset(this->mem.count.ptr, 0x00, this->mem.count.byte);

	hipSetDevice(this->ID);
	hipcub::DeviceScan::ExclusiveSum(nullptr,
								  this->mem.scan.byte,
								  this->mem.lookup[1].ptr,
								  this->mem.lookup[0].ptr,
								  this->mem.lookup[0].count());
	this->mem.scan.ptr = (Count *)this->DM->manualAlloc(this->mem.scan.byte);
	hipMemset(this->mem.scan.ptr, 0x00, this->mem.scan.byte);
}

void ExecutionManager::run()
{
	std::thread([&] {
		auto myInChan  = (this->ID > -1) ? ctx.chan.orderGPU : ctx.chan.orderCPU;
		auto myOutChan = ctx.chan.report[this->ID];

		for (auto & order : *myInChan) {
			std::array<std::array<sp<bchan<DataManager::TxCb>>, 3>, 3> callbacks;

			for (auto & cbrow : callbacks) {
				for (auto & cb : cbrow) {
					cb = makeSp<bchan<DataManager::TxCb>>(1);
				}
			}

			for (uint8_t i = 0; i < callbacks.size(); i++) {
				for (uint8_t j = 0; j < callbacks[i].size(); j++) {
					DataManager::Tx tx;

					tx.idx	  = order[i];
					tx.method = DataManager::Method::ready;
					tx.type	  = (DataManager::Type)j;
					tx.cb	  = callbacks[i][j];
					this->DM->req(tx);
				}
			}

			for (auto & cbrow : callbacks) {
				for (auto & cb : cbrow) {
					for (auto & res : *cb) {
						if (!res.ok) {
							// failed
						}
					}
				}
			}

			Report report;
			report.g3		= order;
			report.deviceID = this->ID;
			report.triangle = 0; // Something Calc

			myOutChan->push(report);
		}
		myOutChan->close();
	}).detach();
}