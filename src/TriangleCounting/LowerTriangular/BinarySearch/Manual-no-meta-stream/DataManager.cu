#include "hip/hip_runtime.h"
#include "DataManager.cuh"
#include "make.cuh"

#include <BuddySystem/BuddySystem.h>
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <fstream>
#include <limits>
#include <mutex>
#include <string>
#include <tuple>
#include <unistd.h>
#include <unordered_map>

static auto genPath(Context & ctx, Key const & k)
{
	auto baseString =
		std::string(ctx.folderPath) + std::to_string(k.idx[0]) + "-" + std::to_string(k.idx[1]);

	fs::path finalPath;
	switch (k.type) {
	case DataType::Row:
		finalPath = fs::path(baseString + ".row");
		break;
	case DataType::Ptr:
		finalPath = fs::path(baseString + ".ptr");
		break;
	case DataType::Col:
		finalPath = fs::path(baseString + ".col");
		break;
	}

	return finalPath;
}

static auto methodDone(Context & ctx, DeviceID myID)
{
	auto in = std::make_shared<bchan<Tx>>(16);
	std::thread([&, myID, in] {
		auto & myCtx = ctx.dataManagerCtx[myID];
		for (auto & tx : *in) {
			MemInfo<Vertex> myInfo = {
				0,
			};
			{
				std::lock_guard<std::mutex> lg(*myCtx.cacheMtx);
				myCtx.cache->at(tx.key).refCnt -= 1;
				// printf("[%2d] %s Done %d -> %d\n", myID, tx.key.print().c_str(),
				// myCtx.cache->at(tx.key).refCnt + 1, myCtx.cache->at(tx.key).refCnt);
			}
			myInfo.ok  = true;
			myInfo.hit = true;
			tx.cb->push(myInfo);
			tx.cb->close();
		}
	}).detach();

	return in;
}

static void tryAllocate(Context &					   ctx,
						Key &						   key,
						DeviceID					   myID,
						MemInfo<Vertex> &			   myInfo,
						std::unique_lock<std::mutex> & ul,
						bool &						   iHaveLock)
{
	auto & myCtx = ctx.dataManagerCtx[myID];

	while (true) {
		myInfo.ptr = (Vertex *)myCtx.buddy->allocate(myInfo.byte);

		if (myInfo.ptr != nullptr) {
			myCtx.cache->insert({key, {myInfo, 1}});
			// printf("[%2d] %s Allc %d -> %d\n", myID, key.print().c_str(),
			// myCtx.cache->at(key).refCnt - 1, myCtx.cache->at(key).refCnt);

			myInfo.ok  = true;
			myInfo.hit = false;

			return;
		} else {
			// allocation failure
			if (myCtx.cache->size() > 0) {
				bool evictSuccess = false;
				while (!evictSuccess) {
					if (!iHaveLock) {
						ul.lock();
						iHaveLock = true;
					}

					for (auto it = myCtx.cache->begin(); it != myCtx.cache->end();) {
						if (it->second.refCnt == 0) {
							myCtx.buddy->deallocate(it->second.info.ptr);
							it			 = myCtx.cache->erase(it);
							evictSuccess = true;
							// printf("[%2d] %s Evict %s\n", myID, key.print().c_str(),
							// it->first.print().c_str());
							break;
						} else {
							++it;
						}
					}

					if (iHaveLock) {
						ul.unlock();
						iHaveLock = false;
					}
				}
			} else {
				throw "Strange Error";
			}
		}
	}
}

static MemInfo<Vertex> requestToReady(Context & ctx, Key & key, DeviceID targetID)
{
	Tx tx;

	tx.key	  = key;
	tx.method = Method::Ready;
	tx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

	ctx.dataManagerCtx[targetID].chan->push(tx);

	MemInfo<Vertex> otherInfo;
	for (auto & info : *tx.cb) { // Code hangs on this line
		otherInfo = info;
	}

	return otherInfo;
}

static auto methodReady(Context & ctx, DeviceID myID)
{
	auto in = std::make_shared<bchan<Tx>>(16);
	std::thread([&, myID, in] {
		for (auto & tx : *in) {
			auto & myCtx = ctx.dataManagerCtx[myID];

			MemInfo<Vertex> myInfo = {
				0,
			};

			std::unique_lock<std::mutex> ul(*myCtx.cacheMtx);

			bool iHaveLock = true;

			if (myCtx.cache->find(tx.key) != myCtx.cache->end()) {
				// printf("[%2d] %s Hit!\n", myID, tx.key.print().c_str());
				myInfo = myCtx.cache->at(tx.key).info;
				myCtx.cache->at(tx.key).refCnt += 1;
				// printf("[%2d] %s Hit  %d -> %d\n", myID, tx.key.print().c_str(),
				// myCtx.cache->at(tx.key).refCnt - 1, myCtx.cache->at(tx.key).refCnt);

				if (iHaveLock) {
					ul.unlock();
					iHaveLock = false;
				}

				myInfo.hit = true;
			} else {
				// printf("[%2d] %s Miss!\n", myID, tx.key.print().c_str());

				myInfo.byte = fs::file_size(genPath(ctx, tx.key));

				tryAllocate(ctx, tx.key, myID, myInfo, ul, iHaveLock);

				// auto	targetID = asktoNeighbor(ctx, tx.key, myID, myInfo);
				DeviceID		targetID; // for debugging
				MemInfo<Vertex> otherInfo;
				// if (targetID != std::numeric_limits<DeviceID>::min()) {
				if (false) { // for debugging
					targetID  = -10;
					otherInfo = requestToReady(ctx, tx.key, targetID);
				} else {
					targetID = myCtx.conn->upstream;
					assert(myID != myCtx.conn->upstream);
					otherInfo = requestToReady(ctx, tx.key, targetID);
				}

				assert(myID >= -1);
				if (myID == -1) {
					// printf("start to read!\n");

					// CPU
					// std::ifstream f(otherInfo.path, std::ios::binary);
					// printf("[%2d] %s fread       SSD[%s]->Host[%p], %ld bytes)\n", myID,
					// tx.key.print().c_str(), otherInfo.path.c_str(), myInfo.ptr, otherInfo.byte);
					auto fp = open64(otherInfo.path.c_str(), O_RDONLY);

					constexpr uint64_t cDef		 = (1L << 30); // chunk Default
					uint64_t		   chunkByte = (myInfo.byte < cDef) ? myInfo.byte : cDef;
					uint64_t		   bytePos	 = 0;
					while (bytePos < myInfo.byte) {
						chunkByte =
							(myInfo.byte - bytePos > chunkByte) ? chunkByte : myInfo.byte - bytePos;
						auto loaded = read(fp, &(((uint8_t *)myInfo.ptr)[bytePos]), chunkByte);
						bytePos += loaded;
					}

					close(fp);
				} else {
					// GPU
					// printf("[%2d] %s hipMemcpy Host[%p]-> GPU[%p], %ld bytes)\n", myID,
					// tx.key.print().c_str(), otherInfo.ptr, myInfo.ptr, otherInfo.byte);
					hipSetDevice(myID);
					hipMemcpyAsync(myInfo.ptr,
									otherInfo.ptr,
									otherInfo.byte,
									hipMemcpyHostToDevice,
									myCtx.stream);
					hipStreamSynchronize(myCtx.stream);
				}

				// printf("[%2d] %s Memcpy/Read complete\n", myID, tx.key.print().c_str());

				if (iHaveLock) {
					ul.unlock();
					iHaveLock = false;
				}

				// Done
				Tx compTx;

				compTx.key	  = tx.key;
				compTx.method = Method::Done;
				compTx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

				ctx.dataManagerCtx[targetID].chan->push(compTx);

				for (auto & res : *compTx.cb) {
				}

				myInfo.hit = false;
			}

			tx.cb->push(myInfo);
			tx.cb->close();
		}
	}).detach();
	return in;
}

void DataManager(Context & ctx, DeviceID myID)
{
	if (myID < -1) {
		// Storage
		std::thread([&, myID] {
			for (auto & tx : *ctx.dataManagerCtx[myID].chan) {
				switch (tx.method) {
				case Method::Ready:
					fiber([&, myID, tx] {
						MemInfo<Vertex> myInfo = {
							0,
						};
						myInfo.ptr	= nullptr;
						myInfo.path = genPath(ctx, tx.key);
						myInfo.byte = fs::file_size(myInfo.path);
						myInfo.ok	= true;
						myInfo.hit	= true;

						tx.cb->push(myInfo);
						tx.cb->close();
					}).detach();
					break;
				case Method::Done:
					fiber([&, myID, tx] {
						MemInfo<Vertex> myInfo = {
							0,
						};

						myInfo.ok = true;

						tx.cb->push(myInfo);
						tx.cb->close();
					}).detach();
					break;
				}
			}
		}).detach();
	} else {
		// Main/GPU Memory
		std::thread([&, myID] {
			auto ReadyChan = methodReady(ctx, myID);
			auto DoneChan  = methodDone(ctx, myID);

			for (auto & tx : *ctx.dataManagerCtx[myID].chan) {
				switch (tx.method) {
				case Method::Ready:
					ReadyChan->push(tx);
					break;
				case Method::Done:
					DoneChan->push(tx);
					break;
				}
			}

			ReadyChan->close();
			DoneChan->close();
		}).detach();
	}
}