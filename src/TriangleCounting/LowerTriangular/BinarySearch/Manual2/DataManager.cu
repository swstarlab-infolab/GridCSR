#include "DataManager.cuh"
#include "make.cuh"

#include <BuddySystem/BuddySystem.h>
#include <hip/hip_runtime.h>
#include <errno.h>
#include <fcntl.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <mutex>
#include <string>
#include <tuple>
#include <unistd.h>
#include <unordered_map>

static auto genPath(Context & ctx, Key const & k)
{
	auto baseString = std::string(ctx.folderPath) + std::to_string(k.idx[0]) + "-" +
					  std::to_string(k.idx[1]) + ".";

	fs::path finalPath;
	switch (k.type) {
	case DataType::Row:
		finalPath = fs::path(baseString + ctx.meta.extension.row);
		break;
	case DataType::Ptr:
		finalPath = fs::path(baseString + ctx.meta.extension.ptr);
		break;
	case DataType::Col:
		finalPath = fs::path(baseString + ctx.meta.extension.col);
		break;
	}

	return finalPath;
}

static auto methodFind(Context & ctx, DeviceID myID)
{
	auto in = std::make_shared<bchan<Tx>>(16);
	std::thread([&, myID, in] {
		auto & myCtx = ctx.dataManagerCtx[myID];

		for (auto & tx : *in) {
			MemInfo<Vertex> myInfo = {
				0,
			};
			{
				std::lock_guard<std::mutex> lg(*myCtx.cacheMtx);
				myInfo.ok = (myCtx.cache->find(tx.key) != myCtx.cache->end());
			}
			myInfo.hit = myInfo.ok;
			tx.cb->push(myInfo);
			tx.cb->close();
		}
	}).detach();

	return in;
}

static auto methodDone(Context & ctx, DeviceID myID)
{
	auto in = std::make_shared<bchan<Tx>>(16);
	std::thread([&, myID, in] {
		auto & myCtx = ctx.dataManagerCtx[myID];
		for (auto & tx : *in) {
			MemInfo<Vertex> myInfo = {
				0,
			};
			{
				std::lock_guard<std::mutex> lg(*myCtx.cacheMtx);
				myCtx.cache->at(tx.key).refCnt -= 1;
				// printf("[%2d] %s Done %d -> %d\n", myID, tx.key.print().c_str(),
				// myCtx.cache->at(tx.key).refCnt + 1, myCtx.cache->at(tx.key).refCnt);
			}
			myInfo.ok  = true;
			myInfo.hit = true;
			tx.cb->push(myInfo);
			tx.cb->close();
		}
	}).detach();

	return in;
}

static void tryAllocate(Context &					   ctx,
						Key &						   key,
						DeviceID					   myID,
						MemInfo<Vertex> &			   myInfo,
						std::unique_lock<std::mutex> & ul,
						bool &						   iHaveLock)
{
	auto & myCtx = ctx.dataManagerCtx[myID];

	while (true) {
		myInfo.ptr = (Vertex *)myCtx.buddy->allocate(myInfo.byte);

		if (myInfo.ptr != nullptr) {
			myCtx.cache->insert({key, {myInfo, 1}});
			// printf("[%2d] %s Allc %d -> %d\n", myID, key.print().c_str(),
			// myCtx.cache->at(key).refCnt - 1, myCtx.cache->at(key).refCnt);

			myInfo.ok  = true;
			myInfo.hit = false;

			return;
		} else {
			// allocation failure
			if (myCtx.cache->size() > 0) {
				bool evictSuccess = false;
				while (!evictSuccess) {
					if (!iHaveLock) {
						ul.lock();
						iHaveLock = true;
					}

					for (auto it = myCtx.cache->begin(); it != myCtx.cache->end();) {
						if (it->second.refCnt == 0) {
							myCtx.buddy->deallocate(it->second.info.ptr);
							it			 = myCtx.cache->erase(it);
							evictSuccess = true;
							// printf("[%2d] %s Evict %s\n", myID, key.print().c_str(),
							// it->first.print().c_str());
							break;
						} else {
							++it;
						}
					}

					if (iHaveLock) {
						ul.unlock();
						iHaveLock = false;
					}
				}
			} else {
				throw "Strange Error";
			}
		}
	}
}

static MemInfo<Vertex> requestToReady(Context & ctx, Key & key, DeviceID targetID)
{
	Tx tx;

	tx.key	  = key;
	tx.method = Method::Ready;
	tx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

	ctx.dataManagerCtx[targetID].chan->push(tx);

	MemInfo<Vertex> otherInfo;
	for (auto & info : *tx.cb) { // Code hangs on this line
		otherInfo = info;
	}

	return otherInfo;
}

static auto methodReady(Context & ctx, DeviceID myID)
{
	auto in = std::make_shared<bchan<Tx>>(16);
	std::thread([&, myID, in] {
		printf("methodReady start at dev %d\n", myID);

		for (auto & tx : *in) {
			auto & myCtx = ctx.dataManagerCtx[myID];

			MemInfo<Vertex> myInfo = {
				0,
			};

			printf("allright dev %d got some request\n", myID);

			std::unique_lock<std::mutex> ul(*myCtx.cacheMtx);
			printf("allright dev %d got mutex\n", myID);

			bool iHaveLock = true;

			if (myCtx.cache->find(tx.key) != myCtx.cache->end()) {
				printf("[%2d] %s Hit!\n", myID, tx.key.print().c_str());
				myInfo = myCtx.cache->at(tx.key).info;
				myCtx.cache->at(tx.key).refCnt += 1;
				printf("[%2d] %s Hit  %d -> %d\n",
					   myID,
					   tx.key.print().c_str(),
					   myCtx.cache->at(tx.key).refCnt - 1,
					   myCtx.cache->at(tx.key).refCnt);

				if (iHaveLock) {
					ul.unlock();
					iHaveLock = false;
				}

				myInfo.hit = true;
			} else {
				printf("[%2d] %s Miss!\n", myID, tx.key.print().c_str());

				myInfo.byte = fs::file_size(genPath(ctx, tx.key));

				tryAllocate(ctx, tx.key, myID, myInfo, ul, iHaveLock);

				auto targetID  = myCtx.conn->upstream;
				auto otherInfo = requestToReady(ctx, tx.key, targetID);

				if (myID == -1) {
					printf("start to read!\n");
					printf("%p\n", myInfo.ptr);
					printf("otherInfo.path.c_str(): %s\n", otherInfo.path.c_str());
					auto fp = open64(otherInfo.path.c_str(), O_RDONLY);
					if (fp < 0) {
						printf("error = %s(%d)\n", strerror(errno), errno);
						exit(EXIT_FAILURE);
					}

					constexpr uint64_t cDef		 = (1L << 30); // chunk Default
					uint64_t		   chunkByte = (myInfo.byte < cDef) ? myInfo.byte : cDef;
					uint64_t		   bytePos	 = 0;
					while (bytePos < myInfo.byte) {
						chunkByte =
							(myInfo.byte - bytePos > chunkByte) ? chunkByte : myInfo.byte - bytePos;

						printf("cbyte bytePos/myInfo.byte = %ld %ld/%ld\n",
							   chunkByte,
							   bytePos,
							   myInfo.byte);
						auto loaded =
							pread(fp, &(((uint8_t *)myInfo.ptr)[bytePos]), chunkByte, bytePos);

						if (loaded < 0) {
							printf("error = %s(%d)\n", strerror(errno), errno);
							exit(EXIT_FAILURE);
						}

						bytePos += loaded;
					}

					close(fp);
				} else {
					// GPU
					// printf("[%2d] %s hipMemcpy Host[%p]-> GPU[%p], %ld bytes)\n", myID,
					// tx.key.print().c_str(), otherInfo.ptr, myInfo.ptr, otherInfo.byte);
					hipSetDevice(myID);
					hipMemcpy(myInfo.ptr, otherInfo.ptr, otherInfo.byte, hipMemcpyHostToDevice);
				}

				printf("[%2d] %s Memcpy/Read complete\n", myID, tx.key.print().c_str());

				if (iHaveLock) {
					ul.unlock();
					iHaveLock = false;
				}

				// Done
				Tx compTx;

				compTx.key	  = tx.key;
				compTx.method = Method::Done;
				compTx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

				ctx.dataManagerCtx[targetID].chan->push(compTx);

				for (auto & res : *compTx.cb) {
				}

				myInfo.hit = false;
			}

			tx.cb->push(myInfo);
			tx.cb->close();
		}
	}).detach();
	return in;
}

void DataManager(Context & ctx, DeviceID myID)
{
	if (myID < -1) {
		// Storage
		std::thread([&, myID] {
			for (auto & tx : *ctx.dataManagerCtx[myID].chan) {
				printf("Storage: I got something!\n");
				switch (tx.method) {
				case Method::Find:
					fiber([&, myID, tx] {
						MemInfo<Vertex> myInfo = {
							0,
						};

						myInfo.hit = true;

						tx.cb->push(myInfo);
						tx.cb->close();
					}).detach();
					break;
				case Method::Ready:
					fiber([&, myID, tx] {
						MemInfo<Vertex> myInfo = {
							0,
						};
						myInfo.ptr	= nullptr;
						myInfo.path = genPath(ctx, tx.key);
						myInfo.byte = fs::file_size(myInfo.path);
						myInfo.ok	= true;
						myInfo.hit	= true;

						tx.cb->push(myInfo);
						tx.cb->close();
					}).detach();
					break;
				case Method::Done:
					fiber([&, myID, tx] {
						MemInfo<Vertex> myInfo = {
							0,
						};

						myInfo.ok = true;

						tx.cb->push(myInfo);
						tx.cb->close();
					}).detach();
					break;
				}
			}
		}).detach();
	} else {
		// Main/GPU Memory
		std::thread([&, myID] {
			auto FindChan  = methodFind(ctx, myID);
			auto ReadyChan = methodReady(ctx, myID);
			auto DoneChan  = methodDone(ctx, myID);

			for (auto & tx : *ctx.dataManagerCtx[myID].chan) {
				switch (tx.method) {
				case Method::Find:
					FindChan->push(tx);
					break;
				case Method::Ready:
					ReadyChan->push(tx);
					break;
				case Method::Done:
					DoneChan->push(tx);
					break;
				}
			}

			FindChan->close();
			ReadyChan->close();
			DoneChan->close();
		}).detach();
	}
}