#include "make.cuh"

#include <sys/mman.h>

int lock_memory(char * addr, size_t size)
{
	unsigned long page_offset, page_size;
	page_size	= sysconf(_SC_PAGE_SIZE);
	page_offset = (unsigned long)addr % page_size;
	addr -= page_offset;
	size += page_offset;
	return mlock(addr, size);
}

int unlock_memory(char * addr, size_t size)
{
	unsigned long page_offset, page_size;
	page_size	= sysconf(_SC_PAGE_SIZE);
	page_offset = (unsigned long)addr % page_size;
	addr -= page_offset;
	size += page_offset;
	return munlock(addr, size);
}

template <>
std::shared_ptr<void> allocCUDA<void>(size_t const byte)
{
	return std::shared_ptr<void>(
		[&] {
			void * p;
			hipMalloc((void **)&p, byte);
			return p;
		}(),
		[](void * p) {
			if (p != nullptr) {
				hipFree(p);
			}
		});
}

template <>
std::shared_ptr<void> allocHost<void>(size_t const byte)
{
	return std::shared_ptr<void>(
		[&] {
			void * p;
			int	   devices = 0;
			hipGetDeviceCount(&devices);
			if (devices) {
				hipHostAlloc((void **)&p, byte, hipHostMallocPortable);
			} else {
				p = malloc(byte);
				lock_memory((char *)p, byte);
			}
			return p;
		}(),
		[byte](void * p) {
			if (p != nullptr) {
				int devices = 0;
				hipGetDeviceCount(&devices);
				if (devices) {
					hipFree(p);
				} else {
					unlock_memory((char *)p, byte);
					free(p);
				}
			}
		});
}