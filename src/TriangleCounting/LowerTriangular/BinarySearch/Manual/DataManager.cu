#include "hip/hip_runtime.h"
#include "DataManager.cuh"
#include "make.cuh"

#include <BuddySystem/BuddySystem.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <limits>
#include <mutex>
#include <string>
#include <tuple>
#include <unistd.h>
#include <unordered_map>

static size_t getFileSize(fs::path const & path)
{
	std::ifstream f;
	f.open(path);
	f.seekg(0, std::ios::end);
	auto const fileSize = f.tellg();
	f.seekg(0, std::ios::beg);
	f.close();
	return fileSize;
}

static auto genPath(Context & ctx, Key const & k)
{
	auto baseString = std::string(ctx.folderPath) + std::to_string(k.idx[0]) + "-" +
					  std::to_string(k.idx[1]) + ".";

	fs::path finalPath;
	switch (k.type) {
	case DataType::Row:
		finalPath = fs::path(baseString + ctx.meta.extension.row);
		break;
	case DataType::Ptr:
		finalPath = fs::path(baseString + ctx.meta.extension.ptr);
		break;
	case DataType::Col:
		finalPath = fs::path(baseString + ctx.meta.extension.col);
		break;
	}

	return finalPath;
}

static void methodFind(Context & ctx, DeviceID myID, Tx tx)
{
	auto & myCtx = ctx.dataManagerCtx[myID];

	MemInfo<Vertex> myInfo = {
		0,
	};
	{
		std::lock_guard<std::mutex> lg(*myCtx.cacheMtx);
		myInfo.ok = (myCtx.cache->find(tx.key) != myCtx.cache->end());
	}
	myInfo.hit = myInfo.ok;
	tx.cb->push(myInfo);
	tx.cb->close();
}

static void methodDone(Context & ctx, int myID, Tx tx)
{
	auto &			myCtx  = ctx.dataManagerCtx[myID];
	MemInfo<Vertex> myInfo = {
		0,
	};
	{
		std::lock_guard<std::mutex> lg(*myCtx.cacheMtx);
		myCtx.cache->at(tx.key).refCnt -= 1;
	}
	myInfo.ok  = true;
	myInfo.hit = true;
	tx.cb->push(myInfo);
	tx.cb->close();
}

static void tryAllocate(Context &					   ctx,
						Key &						   key,
						DeviceID					   myID,
						MemInfo<Vertex> &			   myInfo,
						std::unique_lock<std::mutex> & ul,
						bool &						   iHaveLock)
{
	auto & myCtx = ctx.dataManagerCtx[myID];

	while (true) {
		if (!iHaveLock) {
			ul.lock();
		}

		myInfo.ptr = (Vertex *)myCtx.buddy->allocate(myInfo.byte);

		if (myInfo.ptr != nullptr) {
			myCtx.cache->insert({key, {myInfo, 1}});

			// Early Unlocking for overlap
			// This is insecure because before memcpy completed, it
			// ul.unlock();
			// iHaveLock  = false;
			myInfo.ok  = true;
			myInfo.hit = false;

			// Allocation success. Break
			return;
		} else {
			// allocation failure
			if (myCtx.cache->size() > 0) {
				ul.unlock();
				iHaveLock = false;

				bool evictSuccess = false;
				while (!evictSuccess) {
					ul.lock();
					iHaveLock = true;

					for (auto it = myCtx.cache->begin(); it != myCtx.cache->end();) {
						if (it->second.refCnt == 0) {
							myCtx.buddy->deallocate(it->second.info.ptr);
							it			 = myCtx.cache->erase(it);
							evictSuccess = true;
							break;
						} else {
							++it;
						}
					}

					ul.unlock();
					iHaveLock = false;
				}
			} else {
				ul.unlock();
				iHaveLock = false;

				throw "Strange Error";
			}
		}
	}
}

static DeviceID asktoNeighbor(Context & ctx, Key & key, DeviceID myID, MemInfo<Vertex> & myInfo)
{
	auto & nList = ctx.dataManagerCtx[myID].conn->neighbor;

	if (nList.size() > 0) {
		std::vector<bool>  nSuccess(nList.size());
		std::vector<fiber> waitGroup(nList.size());

		for (size_t i = 0; i < nList.size(); i++) {
			waitGroup[i] = fiber([&, i] {
				Tx tx;
				tx.key	  = key;
				tx.method = Method::Find;
				tx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

				ctx.dataManagerCtx[nList[i]].chan->push(tx);

				for (auto & info : *tx.cb) {
					nSuccess[i] = info.ok;
				}
			});
		}

		for (auto & w : waitGroup) {
			if (w.joinable()) {
				w.join();
			}
		}

		for (size_t i = 0; i < nList.size(); i++) {
			if (nSuccess[i]) {
				return nList[i];
			}
		}
	}

	return std::numeric_limits<DeviceID>::min();
}

static MemInfo<Vertex> requestToReady(Context & ctx, Key & key, DeviceID targetID)
{
	Tx tx;

	tx.key	  = key;
	tx.method = Method::Ready;
	tx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

	ctx.dataManagerCtx[targetID].chan->push(tx);

	MemInfo<Vertex> otherInfo;
	usleep(10);
	for (auto & info : *tx.cb) { // Code hangs on this line
		otherInfo = info;
	}

	return otherInfo;
}

static void methodReady(Context & ctx, int myID, Tx tx)
{
	auto & myCtx = ctx.dataManagerCtx[myID];

	bool			iHaveLock = false;
	MemInfo<Vertex> myInfo	  = {
		   0,
	   };

	std::unique_lock<std::mutex> ul(*myCtx.cacheMtx);
	iHaveLock = true;

	if (myCtx.cache->find(tx.key) != myCtx.cache->end()) {
		// printf("[%2d] %s Hit!\n", myID, tx.key.print().c_str());

		myInfo = myCtx.cache->at(tx.key).info;
		myCtx.cache->at(tx.key).refCnt += 1;

		ul.unlock();
		iHaveLock = false;

		myInfo.hit = true;
	} else {
		// printf("[%2d] %s Miss!\n", myID, tx.key.print().c_str());

		myInfo.byte = getFileSize(genPath(ctx, tx.key));

		tryAllocate(ctx, tx.key, myID, myInfo, ul, iHaveLock);

		// auto	targetID = asktoNeighbor(ctx, tx.key, myID, myInfo);
		DeviceID		targetID; // for debugging
		MemInfo<Vertex> otherInfo;
		// if (targetID != std::numeric_limits<DeviceID>::min()) {
		if (false) { // for debugging
			otherInfo = requestToReady(ctx, tx.key, targetID);
		} else {
			targetID = myCtx.conn->upstream;
			assert(myID != myCtx.conn->upstream);
			otherInfo = requestToReady(ctx, tx.key, targetID);
		}

		assert(myID >= -1);
		if (myID == -1) {
			// printf("start to read!\n");
			// CPU
			std::ifstream f(otherInfo.path, std::ios::binary);
			f.read((char *)myInfo.ptr, otherInfo.byte);
			// printf("[%2d] %s fread       SSD[%s]->Host[%p], %ld bytes)\n", myID,
			// tx.key.print().c_str(), otherInfo.path.c_str(), myInfo.ptr, otherInfo.byte);
			f.close();
		} else {
			// GPU
			// printf("[%2d] %s hipMemcpy Host[%p]-> GPU[%p], %ld bytes)\n", myID,
			// tx.key.print().c_str(), otherInfo.ptr, myInfo.ptr, otherInfo.byte);
			hipSetDevice(myID);
			hipMemcpy(myInfo.ptr, otherInfo.ptr, otherInfo.byte, hipMemcpyHostToDevice);
		}

		if (iHaveLock) {
			ul.unlock();
			iHaveLock = false;
		}

		// Done
		Tx compTx;

		compTx.key	  = tx.key;
		compTx.method = Method::Done;
		compTx.cb	  = std::make_shared<bchan<MemInfo<Vertex>>>(2);

		ctx.dataManagerCtx[targetID].chan->push(compTx);

		for (auto & res : *compTx.cb) {
		}

		myInfo.hit = false;
	}

	myInfo.ok = true;
	tx.cb->push(myInfo);
	tx.cb->close();
	return;
}

void DataManager(Context & ctx, int myID)
{
	if (myID < -1) {
		// Storage
		std::thread([&, myID] {
			for (auto & tx : *ctx.dataManagerCtx[myID].chan) {
				switch (tx.method) {
				case Method::Find:
					std::thread([&, myID, tx] {
						MemInfo<Vertex> myInfo = {
							0,
						};

						myInfo.hit = true;

						tx.cb->push(myInfo);
						tx.cb->close();
					}).detach();
					break;
				case Method::Ready:
					std::thread([&, myID, tx] {
						MemInfo<Vertex> myInfo = {
							0,
						};
						myInfo.ptr	= nullptr;
						myInfo.path = genPath(ctx, tx.key);
						myInfo.byte = getFileSize(myInfo.path);
						myInfo.ok	= true;
						myInfo.hit	= true;

						tx.cb->push(myInfo);
						tx.cb->close();
					}).detach();
					break;
				case Method::Done:
					std::thread([&, myID, tx] {
						MemInfo<Vertex> myInfo = {
							0,
						};

						myInfo.ok = true;

						tx.cb->push(myInfo);
						tx.cb->close();
					}).detach();
					break;
				}
			}
		}).detach();
	} else {
		// Main/GPU Memory
		std::thread([&, myID] {
			for (auto & tx : *ctx.dataManagerCtx[myID].chan) {
				switch (tx.method) {
				case Method::Find:
					std::thread([&, myID, tx] { methodFind(ctx, myID, tx); }).detach();
					break;
				case Method::Ready:
					std::thread([&, myID, tx] { methodReady(ctx, myID, tx); }).detach();
					break;
				case Method::Done:
					std::thread([&, myID, tx] { methodDone(ctx, myID, tx); }).detach();
					break;
				}
			}
		}).detach();
	}
}