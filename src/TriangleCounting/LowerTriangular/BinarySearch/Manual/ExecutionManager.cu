#include "hip/hip_runtime.h"
#include "ExecutionManager.cuh"
#include "make.cuh"
#include "type.cuh"

#include <BuddySystem/BuddySystem.h>
#include <GridCSR/CUDA/Kernel.cuh>
#include <array>
#include <chrono>
#include <cub/device/device_scan.cuh>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <memory>
#include <thread>

#define CUDACHECK()                        \
	do {                                   \
		auto e = hipGetLastError();       \
		if (e) {                           \
			printf("%s:%d, %s(%d), %s\n",  \
				   __FILE__,               \
				   __LINE__,               \
				   hipGetErrorName(e),    \
				   e,                      \
				   hipGetErrorString(e)); \
			hipDeviceReset();             \
			exit(EXIT_FAILURE);            \
		}                                  \
	} while (false)

static __global__ void genLookupTemp(Grid const g, Lookup * luTemp)
{
	for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < g[0].count();
		 i += gridDim.x * blockDim.x) {
		luTemp[g[0][i]] = g[1][i + 1] - g[1][i];
	}
}

static __global__ void resetLookupTemp(Grid const g, Lookup * luTemp)
{
	for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < g[0].count();
		 i += gridDim.x * blockDim.x) {
		luTemp[g[0][i]] = 0;
	}
}

static __global__ void
kernel(Grids const g, Lookup const * lookup0, Lookup const * lookup2, Count * count)
{
	Count mycount = 0;

	__shared__ int SHARED[1024];

	for (uint32_t g1row_iter = blockIdx.x; g1row_iter < g[1][0].count(); g1row_iter += gridDim.x) {

		// This makes huge difference!!!
		// Without "Existing Row" information: loop all 2^24 and check it all
		// With "Existing Row" information: extremely faster than without-version
		auto const g1row = g[1][0][g1row_iter];

		if (lookup2[g1row] == lookup2[g1row + 1]) {
			continue;
		}

		auto const g1col_idx_s = g[1][1][g1row_iter];
		auto const g1col_idx_e = g[1][1][g1row_iter + 1];

		// variable for binary tree intersection
		auto const g1col_length = g1col_idx_e - g1col_idx_s;

		auto const g2col_s = lookup2[g1row], g2col_e = lookup2[g1row + 1];

		for (uint32_t g2col_idx = g2col_s; g2col_idx < g2col_e; g2col_idx += blockDim.x) {
			SHARED[threadIdx.x] =
				(g2col_idx + threadIdx.x < g2col_e) ? (int)g[2][2][g2col_idx + threadIdx.x] : -1;

			__syncthreads();

			for (uint32_t s = 0; s < blockDim.x; s++) {
				int const g2col = SHARED[s];
				if (g2col == -1) {
					break;
				}
				if (lookup0[g2col] == lookup0[g2col + 1]) {
					continue;
				}

				auto const g0col_idx_s = lookup0[g2col], g0col_idx_e = lookup0[g2col + 1];

				// variable for binary tree intersection
				auto const g0col_length = g0col_idx_e - g0col_idx_s;

				if (g1col_length >= g0col_length) {
					for (uint32_t g0col_idx = g0col_idx_s + threadIdx.x; g0col_idx < g0col_idx_e;
						 g0col_idx += blockDim.x) {
						GridCSR::CUDA::BinarySearchIntersection(
							&g[1][2][g1col_idx_s], g1col_length, g[0][2][g0col_idx], &mycount);
					}
				} else {
					for (uint32_t g1col_idx = g1col_idx_s + threadIdx.x; g1col_idx < g1col_idx_e;
						 g1col_idx += blockDim.x) {
						GridCSR::CUDA::BinarySearchIntersection(
							&g[0][2][g0col_idx_s], g0col_length, g[1][2][g1col_idx], &mycount);
					}
				}
			}
			__syncthreads();
		}
	}

	for (uint8_t offset = 16; offset > 0; offset >>= 1) {
		mycount += __shfl_down_sync(0xFFFFFFFF, mycount, offset);
	}

	if ((threadIdx.x & 31) == 0) {
		atomicAdd(count, mycount);
	}
}

static Count launchKernel(Context & ctx, DeviceID myID, Grids & G)
{
	auto & myCtx   = ctx.executionManagerCtx[myID];
	auto & blocks  = ctx.setting[1];
	auto & threads = ctx.setting[2];

	// if (!(G[0][0].byte && G[1][0].byte && G[2][0].byte)) { return 0; }

	// hipStream_t stream;

	hipSetDevice(myID);
	hipMemset(myCtx.lookup.temp.ptr, 0, myCtx.lookup.temp.byte);
	hipMemset(myCtx.lookup.G0.ptr, 0, myCtx.lookup.G0.byte);
	hipMemset(myCtx.lookup.G2.ptr, 0, myCtx.lookup.G2.byte);
	hipMemset(myCtx.count.ptr, 0, myCtx.count.byte);

	// hipStreamCreate(&stream);
	// CUDACHECK();
	hipSetDevice(myID);
	genLookupTemp<<<blocks, threads>>>(G[0], myCtx.lookup.temp.ptr);
	hipDeviceSynchronize();

	hipSetDevice(myID);
	hipcub::DeviceScan::ExclusiveSum(myCtx.cub.ptr,
								  myCtx.cub.byte,
								  myCtx.lookup.temp.ptr,
								  myCtx.lookup.G0.ptr,
								  myCtx.lookup.G0.count());
	hipDeviceSynchronize();

	hipSetDevice(myID);
	resetLookupTemp<<<blocks, threads>>>(G[0], myCtx.lookup.temp.ptr);
	hipDeviceSynchronize();

	hipSetDevice(myID);
	genLookupTemp<<<blocks, threads>>>(G[2], myCtx.lookup.temp.ptr);
	hipDeviceSynchronize();

	hipSetDevice(myID);
	hipcub::DeviceScan::ExclusiveSum(myCtx.cub.ptr,
								  myCtx.cub.byte,
								  myCtx.lookup.temp.ptr,
								  myCtx.lookup.G2.ptr,
								  myCtx.lookup.G2.count());
	hipDeviceSynchronize();

	hipSetDevice(myID);
	resetLookupTemp<<<blocks, threads>>>(G[2], myCtx.lookup.temp.ptr);
	hipDeviceSynchronize();

	hipSetDevice(myID);
	kernel<<<blocks, threads>>>(G, myCtx.lookup.G0.ptr, myCtx.lookup.G2.ptr, myCtx.count.ptr);

	// hipSetDevice(myID);
	// edgeCount<<<blocks, threads>>>(G, myCtx.count.ptr);

	Count cnt = 0;
	hipSetDevice(myID);
	// hipMemcpyAsync(&cnt, myCtx.count.ptr, sizeof(Count), hipMemcpyDeviceToHost, stream);
	hipMemcpy(&cnt, myCtx.count.ptr, sizeof(Count), hipMemcpyDeviceToHost);
	CUDACHECK();

	// hipStreamSynchronize(stream); CUDACHECK();
	// hipDeviceSynchronize(); CUDACHECK();
	// hipStreamDestroy(stream); CUDACHECK();

	return cnt;
}
// Under construction...

static void ExecutionGPU(Context &							   ctx,
						 DeviceID							   myID,
						 std::shared_ptr<bchan<Command>>	   in,
						 std::shared_ptr<bchan<CommandResult>> out)
{
	using DataTxCallback = bchan<MemInfo<Vertex>>;

	size_t hitCount = 0, missCount = 0;

	for (auto & req : *in) {
		// PREPARE
		auto start = std::chrono::system_clock::now();

		Grids								memInfo;
		std::array<std::array<fiber, 3>, 3> waitGroup;
		for (uint32_t i = 0; i < 3; i++) {
			for (uint32_t type = 0; type < 3; type++) {
				waitGroup[i][type] = fiber([&, myID, i, type] {
					auto callback = std::make_shared<DataTxCallback>(2);

					Tx tx;
					tx.method = Method::Ready;
					tx.key	  = {req.gidx[i], (DataType)(type)};
					tx.cb	  = callback;

					ctx.dataManagerCtx[myID].chan->push(tx);

					for (auto & cbres : *callback) {
						memInfo[i][type] = cbres;
					}
				});
			}
		}

		// Must wait all memory info
		for (auto & row : waitGroup) {
			for (auto & w : row) {
				if (w.joinable()) {
					w.join();
				}
			}
		}

		for (auto & row : memInfo) {
			for (auto & i : row) {
				if (i.hit) {
					hitCount++;
				} else {
					missCount++;
				}
			}
		}

		// LAUNCH
		auto tri = launchKernel(ctx, myID, memInfo);

		/*
				printf("Kernel End:\n"
					   "(%d,%d):[%s,%s,%s]\n"
					   "(%d,%d):[%s,%s,%s]\n"
					   "(%d,%d):[%s,%s,%s]\n",
					   req.gidx[0][0],
					   req.gidx[0][1],
					   memInfo[0][0].print().c_str(),
					   memInfo[0][1].print().c_str(),
					   memInfo[0][2].print().c_str(),
					   req.gidx[1][0],
					   req.gidx[1][1],
					   memInfo[1][0].print().c_str(),
					   memInfo[1][1].print().c_str(),
					   memInfo[1][2].print().c_str(),
					   req.gidx[2][0],
					   req.gidx[2][1],
					   memInfo[2][0].print().c_str(),
					   memInfo[2][1].print().c_str(),
					   memInfo[2][2].print().c_str());
					   */

		auto end = std::chrono::system_clock::now();

		// RELEASE MEMORY
		for (uint32_t i = 0; i < 3; i++) {
			for (uint32_t type = 0; type < 3; type++) {
				waitGroup[i][type] = fiber([&, myID, i, type] {
					auto callback = std::make_shared<DataTxCallback>(2);

					Tx tx;
					tx.method = Method::Done;
					tx.key	  = {req.gidx[i], (DataType)(type)};
					tx.cb	  = callback;

					ctx.dataManagerCtx[myID].chan->push(tx);

					for (auto & cbres : *callback) {
						memInfo[i][type] = cbres;
					}
				});
			}
		}

		for (auto & row : waitGroup) {
			for (auto & w : row) {
				if (w.joinable()) {
					w.join();
				}
			}
		}

		// CALLBACK RESPONSE
		CommandResult res;
		res.gidx		= req.gidx;
		res.deviceID	= myID;
		res.triangle	= tri;
		res.elapsedTime = std::chrono::duration<double>(end - start).count();

		out->push(res);
	}

	ctx.dataManagerCtx[myID].chan->close();
	out->close();

	printf("HIT: %ld, MISS: %ld, HIT/TOTAL: %lf\n",
		   hitCount,
		   missCount,
		   double(hitCount) / double(hitCount + missCount));
}

std::shared_ptr<bchan<CommandResult>>
ExecutionManager(Context & ctx, int myID, std::shared_ptr<bchan<Command>> in)
{
	// auto out = make<bchan<CommandResult>>(1 << 4);
	auto out = std::make_shared<bchan<CommandResult>>(1 << 4);
	// prepare channels
	if (myID < -1) {
		// No operation
	} else if (myID == -1) {
		// std::thread([&, myID, in, out] { ExecutionCPU(ctx, myID, in, out); }).detach();
	} else {
		std::thread([&, myID, in, out] { ExecutionGPU(ctx, myID, in, out); }).detach();
	}

	return out;
}