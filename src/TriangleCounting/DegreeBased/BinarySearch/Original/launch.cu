#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <chrono>
#include <array>

#include "tc.h"
#include "device-setting.cuh"

#include <cmath>

__device__ static uint32_t ulog2floor(uint32_t x) {
    uint32_t r, q;
    r = (x > 0xFFFF) << 4; x >>= r;
    q = (x > 0xFF  ) << 3; x >>= q; r |= q;
    q = (x > 0xF   ) << 2; x >>= q; r |= q;
    q = (x > 0x3   ) << 1; x >>= q; r |= q;
                                   
    return (r | (x >> 1));
}

__device__ static void intersection(
    GridCSR::Vertex const * Arr,
    uint32_t const ArrLen,
    GridCSR::Vertex const candidate,
    count_t * count)
{
    //auto const maxLevel = uint32_t(ceil(log2(ArrLen + 1))) - 1;
    // ceil(log2(a)) == floor(log2(a-1))+1
    auto const maxLevel = ulog2floor(ArrLen);

    int now = (ArrLen - 1) >> 1;

    for (uint32_t level = 0; level <= maxLevel; level++) {
        auto const movement = 1 << (maxLevel - level - 1);

        if (now < 0) {
            now += movement;
        } else if (ArrLen <= now) {
            now -= movement;
        } else {
            if (Arr[now] < candidate) {
                now += movement;
            } else if (candidate < Arr[now]) {
                now -= movement;
            } else {
                (*count)++;
                break;
            }
        }
    }
}

__device__ static int binarySearchPosition(
    GridCSR::Vertex const * Arr,
    uint32_t const ArrLen,
    GridCSR::Vertex const candidate)
{
    //auto const maxLevel = uint32_t(ceil(log2(ArrLen + 1))) - 1;
    // ceil(log2(a)) == floor(log2(a-1))+1
    auto const maxLevel = ulog2floor(ArrLen);

    int now = (ArrLen - 1) >> 1;

    for (uint32_t level = 0; level <= maxLevel; level++) {
        auto const movement = 1 << (maxLevel - level - 1);

        if (now < 0) {
            now += movement;
        } else if (ArrLen <= now) {
            now -= movement;
        } else {
            if (Arr[now] < candidate) {
                now += movement;
            } else if (candidate < Arr[now]) {
                now -= movement;
            } else {
                return now;
            }
        }
    }

    return -1;
}

struct kernelParameter {
    struct {
        struct {
            GridCSR::Vertex *row, *ptr, *col;
            GridCSR::Vertex rows, ptrs, cols;
        } p, a, b;
    } G;

    count_t *count;
};

__global__ static void kernel(kernelParameter kp)
{
    count_t mycount = 0;

    auto const & G = kp.G;
    __shared__ int SHARED[1024];

    for (GridCSR::Vertex prowIter = blockIdx.x; prowIter < G.p.rows; prowIter+=gridDim.x) {
        int const apos = binarySearchPosition(G.a.row, G.a.rows, G.p.row[prowIter]);
        if (apos == -1) { continue; }
        int const alen = G.a.ptr[apos+1] - G.a.ptr[apos];

        auto const Gpptr_s = G.p.ptr[prowIter];
        auto const Gpptr_e = G.p.ptr[prowIter+1];

        for (GridCSR::Vertex pcolIter = Gpptr_s; pcolIter < Gpptr_e; pcolIter+=blockDim.x) {
            SHARED[threadIdx.x]
                    = (pcolIter + threadIdx.x < Gpptr_e) ?
                        binarySearchPosition(G.b.row, G.b.rows, G.p.col[pcolIter+threadIdx.x]) : -2;

            __syncthreads();

            for (uint32_t t = 0; t < blockDim.x; t++) {
                int const bpos = SHARED[t];
                if (bpos == -2) { break;} // very important for runtime. (x2 speedup)
                if (bpos == -1) { continue; }

                int const blen = G.b.ptr[bpos+1] - G.b.ptr[bpos];

                if (alen > blen) {
                    for (GridCSR::Vertex bcolIter = G.b.ptr[bpos]+threadIdx.x; bcolIter < G.b.ptr[bpos+1]; bcolIter+=blockDim.x) {
                        intersection(&G.a.col[G.a.ptr[apos]], alen, G.b.col[bcolIter], &mycount);
                    }
                } else {
                    for (GridCSR::Vertex acolIter = G.a.ptr[apos]+threadIdx.x; acolIter < G.a.ptr[apos+1]; acolIter+=blockDim.x) {
                        intersection(&G.b.col[G.b.ptr[bpos]], blen, G.a.col[acolIter], &mycount);
                    }
                }
            }

            __syncthreads();
        }
    }

    for (uint8_t offset = 16; offset > 0; offset >>= 1) {
		mycount += __shfl_down_sync(0xFFFFFFFF, mycount, offset);
	}

	if ((threadIdx.x & 31) == 0) { atomicAdd(kp.count, mycount); }
}

void launch(std::vector<device_setting_t> & dev) {
    std::vector<count_t> globalCount(dev.size() * dev.front().gpu.setting.stream.size());

    size_t streamIndex = 0;
    size_t deviceIndex = 0;

    auto next = [&dev, &deviceIndex, &streamIndex]() {
        streamIndex++;
        if (streamIndex / dev[deviceIndex].gpu.setting.stream.size()) {
            streamIndex = 0;
            deviceIndex++;
            if (deviceIndex / dev.size()) {
                deviceIndex = 0;
            }
        }
    };

    auto const gridCount = dev.front().mem.graph_meta.info.count.row;
    auto const gridWidth = dev.front().mem.graph_meta.info.width.row;

    auto start = std::chrono::system_clock::now();


    for (size_t row = 0; row < gridCount; row++) {
        for (size_t col = 0; col < gridCount; col++) {
            auto & d = dev[deviceIndex];
            hipSetDevice(d.gpu.meta.index); CUDACHECK();

            auto & mem = d.mem.stream[streamIndex];
            auto & setting = d.gpu.setting;

            auto const & Gp = d.mem.graph[row][col];

            for (size_t col2 = 0; col2 < gridCount; col2++) {
                auto const & Ga = d.mem.graph[row][col2];
                auto const & Gb = d.mem.graph[col][col2];

                kernelParameter kp;

                kp.G.p.row = Gp.row.ptr; kp.G.p.ptr = Gp.ptr.ptr; kp.G.p.col = Gp.col.ptr;
                kp.G.a.row = Ga.row.ptr; kp.G.a.ptr = Ga.ptr.ptr; kp.G.a.col = Ga.col.ptr;
                kp.G.b.row = Gb.row.ptr; kp.G.b.ptr = Gb.ptr.ptr; kp.G.b.col = Gb.col.ptr;

                kp.G.p.rows = Gp.row.count; kp.G.p.ptrs = Gp.ptr.count; kp.G.p.cols = Gp.col.count; 
                kp.G.a.rows = Ga.row.count; kp.G.a.ptrs = Ga.ptr.count; kp.G.a.cols = Ga.col.count; 
                kp.G.b.rows = Gb.row.count; kp.G.b.ptrs = Gb.ptr.count; kp.G.b.cols = Gb.col.count; 

                kp.count = mem.count.ptr;

                kernel<<<setting.block, setting.thread, 0, setting.stream[streamIndex]>>>(kp);

            }
            next();
        }
    }

    for (auto & d : dev) {
        hipSetDevice(d.gpu.meta.index); CUDACHECK();
        for (size_t i = 0; i < d.gpu.setting.stream.size(); i++) {
            d.mem.stream[i].count.copy_d2h_async(&globalCount[d.gpu.setting.stream.size() * d.gpu.meta.index + i], d.gpu.setting.stream[i]); CUDACHECK();
        }
    }

    for (auto & d : dev) {
        hipSetDevice(d.gpu.meta.index); CUDACHECK();
        for (size_t i = 0; i < d.gpu.setting.stream.size(); i++) {
            hipStreamSynchronize(d.gpu.setting.stream[i]); CUDACHECK();
        }
    }

    for (size_t i = 1; i < globalCount.size(); i++) {
        globalCount.front() += globalCount[i];
    }

    std::chrono::duration<double> elapsed = std::chrono::system_clock::now() - start;
    std::cout << globalCount.front() << "," << elapsed.count() << std::endl;
}